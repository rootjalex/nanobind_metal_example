#include "utils.h"

#include <hip/hip_runtime.h>

// Wrapper function for hipFree
void cudaFreeWrapper(void* ptr) noexcept {
    // Can't do this due to `noexcept`
    // hipError_t error = hipFree(ptr);
    // if (error != hipSuccess) {
    //     throw std::runtime_error(hipGetErrorString(error));
    // }
    hipFree(ptr);
}
